#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <tuple>
#include <omp.h>
#include <dirent.h>

#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)
int THREADS_PER_BLOCK = 256;
int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

//detect extrema (deprecated)
template <typename coord_t, typename real_t>
__global__ void find_extrema_shfl(const coord_t* d_multiChannelIndex, const real_t* d_ProjectSignals, coord_t* d_sparseMaxIndex, real_t* d_sparseMaxValue, coord_t* d_sparseMinIndex, real_t* d_sparseMinValue, coord_t* d_sparseMaxFlag, coord_t* d_sparseMinFlag, size_t SignalLength) {

    int channelIndex = blockIdx.y;
    int channelElementsIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int allElementsIndex = blockIdx.y * SignalLength + blockIdx.x * blockDim.x + threadIdx.x;

    int warpFlag = channelElementsIndex / 32;

    if ((channelElementsIndex - 2 * warpFlag) < SignalLength)
    {
        real_t value = d_ProjectSignals[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];
        coord_t coord = d_multiChannelIndex[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];

        real_t up = __shfl_up_sync(0xffffffff, value, 1);
        real_t down = __shfl_down_sync(0xffffffff, value, 1);
        if (value > up && value > down)
        {
            d_sparseMaxIndex[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)] = coord;
            d_sparseMaxValue[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)] = value;
            d_sparseMaxFlag[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)] = 1;
        }


        if (value < up && value < down)
        {
            d_sparseMinIndex[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)] = coord;
            d_sparseMinValue[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)] = value;
            d_sparseMinFlag[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)] = 1;
        }
        // for temporary, set edge points as it were 
        if (channelElementsIndex == 0)
        {
            d_sparseMaxIndex[blockIdx.y * SignalLength] = d_multiChannelIndex[blockIdx.y * SignalLength];
            d_sparseMaxValue[blockIdx.y * SignalLength] = d_ProjectSignals[blockIdx.y * SignalLength];
            d_sparseMaxFlag[blockIdx.y * SignalLength] = 1;

            d_sparseMinIndex[blockIdx.y * SignalLength] = d_multiChannelIndex[blockIdx.y * SignalLength];
            d_sparseMinValue[blockIdx.y * SignalLength] = d_ProjectSignals[blockIdx.y * SignalLength];
            d_sparseMinFlag[blockIdx.y * SignalLength] = 1;
        }

        if ((channelElementsIndex - 2 * warpFlag) == (SignalLength - 1))
        {
            d_sparseMaxIndex[blockIdx.y * SignalLength + (SignalLength - 1)] = d_multiChannelIndex[blockIdx.y * SignalLength + (SignalLength - 1)];
            d_sparseMaxValue[blockIdx.y * SignalLength + (SignalLength - 1)] = d_ProjectSignals[blockIdx.y * SignalLength + (SignalLength - 1)];
            d_sparseMaxFlag[blockIdx.y * SignalLength + (SignalLength - 1)] = 1;

            d_sparseMinIndex[blockIdx.y * SignalLength + (SignalLength - 1)] = d_multiChannelIndex[blockIdx.y * SignalLength + (SignalLength - 1)];
            d_sparseMinValue[blockIdx.y * SignalLength + (SignalLength - 1)] = d_ProjectSignals[blockIdx.y * SignalLength + (SignalLength - 1)];
            d_sparseMinFlag[blockIdx.y * SignalLength + (SignalLength - 1)] = 1;
        }
    }

}

template <typename coord_t, typename real_t>
__global__ void find_extrema_shfl_max(const coord_t* d_multiChannelIndex, const real_t* d_ProjectSignals, coord_t* d_sparseMaxFlag, size_t SignalLength) {

    int channelIndex = blockIdx.y;
    int channelElementsIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int allElementsIndex = blockIdx.y * SignalLength + blockIdx.x * blockDim.x + threadIdx.x;

    int warpFlag = channelElementsIndex / 32;

    if ((channelElementsIndex - 2 * warpFlag) < SignalLength)
    {
        real_t value = d_ProjectSignals[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];
        coord_t coord = d_multiChannelIndex[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];

        real_t up = __shfl_up_sync(0xffffffff, value, 1);
        real_t down = __shfl_down_sync(0xffffffff, value, 1);
        if (value > up && value > down)
        {
            d_sparseMaxFlag[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)] = 1;
        }

        // for temporary, set edge points as it were 
        if (channelElementsIndex == 0)
        {
            d_sparseMaxFlag[blockIdx.y * SignalLength] = 1;
            d_sparseMaxFlag[blockIdx.y * SignalLength + SignalLength - 1] = 1;
        }

        //deprecated
        //if ((channelElementsIndex - 2 * warpFlag) == (SignalLength - 1))
        //{
        //    d_sparseMaxFlag[blockIdx.y * SignalLength + (SignalLength - 1)] = 1;
        //}
    }
}

template <typename coord_t, typename real_t>
__global__ void find_extrema_shfl_min(const coord_t* d_multiChannelIndex, const real_t* d_ProjectSignals, coord_t* d_sparseMinFlag, size_t SignalLength) {

    int channelIndex = blockIdx.y;
    int channelElementsIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int allElementsIndex = blockIdx.y * SignalLength + blockIdx.x * blockDim.x + threadIdx.x;

    int warpFlag = channelElementsIndex / 32;

    if ((channelElementsIndex - 2 * warpFlag) < SignalLength)
    {
        real_t value = d_ProjectSignals[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];
        coord_t coord = d_multiChannelIndex[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];

        real_t up = __shfl_up_sync(0xffffffff, value, 1);
        real_t down = __shfl_down_sync(0xffffffff, value, 1);

        if (value < up && value < down)
        {
            d_sparseMinFlag[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)] = 1;
        }
        // for temporary, set edge points as it were 
        if (channelElementsIndex == 0)
        {
            d_sparseMinFlag[blockIdx.y * SignalLength] = 1;
            d_sparseMinFlag[blockIdx.y * SignalLength + SignalLength - 1] = 1;
        }
    }
}

__global__ void prescan_arbitrary(int* output, int* input, int n, int powerOfTwo)
{
    extern __shared__ int temp[];// allocated on invocation
    int threadID = threadIdx.x;

    int ai = threadID;
    int bi = threadID + (n / 2);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);


    if (threadID < n) {
        temp[ai + bankOffsetA] = input[ai];
        temp[bi + bankOffsetB] = input[bi];
    }
    else {
        temp[ai + bankOffsetA] = 0;
        temp[bi + bankOffsetB] = 0;
    }


    int offset = 1;
    for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (threadID == 0) {
        temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0; // clear the last element
    }

    for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    if (threadID < n) {
        output[ai] = temp[ai + bankOffsetA];
        output[bi] = temp[bi + bankOffsetB];
    }
}

__global__ void prescan_large(int* output, int* input, int n, int* sums) {
    extern __shared__ int temp[];

    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * n;

    int ai = threadID;
    int bi = threadID + (n / 2);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    temp[ai + bankOffsetA] = input[blockOffset + ai];
    temp[bi + bankOffsetB] = input[blockOffset + bi];

    int offset = 1;
    for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    __syncthreads();


    if (threadID == 0) {
        sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
        temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
    }

    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    output[blockOffset + ai] = temp[ai + bankOffsetA];
    output[blockOffset + bi] = temp[bi + bankOffsetB];
}

__global__ void add(int* output, int length, int* n) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * length;

    output[blockOffset + threadID] += n[blockID];
}

__global__ void add(int* output, int length, int* n1, int* n2) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * length;

    output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}

__global__ void find_extrema_shfl_max_4matlab(const int* d_multiChannelIndex, const float* d_ProjectSignals, int* d_sparseMaxFlag, int SignalLength) {

    int channelIndex = blockIdx.y;
    int channelElementsIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int allElementsIndex = blockIdx.y * SignalLength + blockIdx.x * blockDim.x + threadIdx.x;

    int warpFlag = channelElementsIndex / 32;

    if ((channelElementsIndex - 2 * warpFlag) < SignalLength)
    {
        float value = d_ProjectSignals[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];
        int coord = d_multiChannelIndex[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];

        float up = __shfl_up_sync(0xffffffff, value, 1);
        float down = __shfl_down_sync(0xffffffff, value, 1);
        if (value > up && value > down)
        {
            d_sparseMaxFlag[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)] = 1;
        }

        // for temporary, set edge points as it were 
        if (channelElementsIndex == 0)
        {
            d_sparseMaxFlag[blockIdx.y * SignalLength] = 1;
            d_sparseMaxFlag[blockIdx.y * SignalLength + SignalLength - 1] = 1;
        }

        //deprecated
        //if ((channelElementsIndex - 2 * warpFlag) == (SignalLength - 1))
        //{
        //    d_sparseMaxFlag[blockIdx.y * SignalLength + (SignalLength - 1)] = 1;
        //}
    }
}

void scanLargeDeviceArray(int* d_out, int* d_in, int length, bool bcao, int* d_sums, int* d_incr);
void scanSmallDeviceArray(int* d_out, int* d_in, int length, bool bcao);
void scanLargeEvenDeviceArray(int* d_out, int* d_in, int length, bool bcao, int* d_sums, int* d_incr);

int nextPowerOfTwo(int x) {
    int power = 1;
    while (power < x) {
        power *= 2;
    }
    return power;
}

void scanLargeDeviceArray(int* d_out, int* d_in, int length, bool bcao, int* d_sums, int* d_incr) {
    int remainder = length % (ELEMENTS_PER_BLOCK);
    if (remainder == 0) {
        scanLargeEvenDeviceArray(d_out, d_in, length, bcao, d_sums, d_incr);
    }
    else {
        // perform a large scan on a compatible multiple of elements
        int lengthMultiple = length - remainder;
        scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple, bcao, d_sums, d_incr);

        // scan the remaining elements and add the (inclusive) last element of the large scan to this
        int* startOfOutputArray = &(d_out[lengthMultiple]);
        scanSmallDeviceArray(startOfOutputArray, &(d_in[lengthMultiple]), remainder, bcao);

        add << <1, remainder >> > (startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
    }
}

void scanSmallDeviceArray(int* d_out, int* d_in, int length, bool bcao) {
    int powerOfTwo = nextPowerOfTwo(length);

    if (bcao) {
        prescan_arbitrary << <1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int) >> > (d_out, d_in, length, powerOfTwo);
    }
    else {
        //prescan_arbitrary_unoptimized << <1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int) >> > (d_out, d_in, length, powerOfTwo);
    }
}

void scanLargeEvenDeviceArray(int* d_out, int* d_in, int length, bool bcao, int* d_sums, int* d_incr) {
    const int blocks = length / ELEMENTS_PER_BLOCK;
    const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

    if (bcao) {
        prescan_large << <blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize >> > (d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
    }
    else {
        //prescan_large_unoptimized << <blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize >> > (d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
    }

    const int sumsArrThreadsNeeded = (blocks + 1) / 2;
    if (sumsArrThreadsNeeded > THREADS_PER_BLOCK) {
        // perform a large scan on the sums arr
        scanLargeDeviceArray(d_incr, d_sums, blocks, bcao, d_sums, d_incr);
    }
    else {
        // only need one block to scan sums arr so can use small scan
        scanSmallDeviceArray(d_incr, d_sums, blocks, bcao);
    }

    add << <blocks, ELEMENTS_PER_BLOCK >> > (d_out, ELEMENTS_PER_BLOCK, d_incr);

}

template <typename coord_t, typename real_t>
__global__ void select_extrema_max(coord_t* d_sparseMaxFlag, real_t* d_current, coord_t* d_multiDirVecChanSignalIndex, coord_t* d_MaxScanResult,
    real_t* d_compactMaxValue, coord_t* d_compactMaxIndex, size_t SignalLength, size_t SignalDim, size_t NumDirVector, coord_t* d_num_extrema_max)
{
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pointsIdx < SignalLength)
    {
        real_t currentValue = d_current[signalDimIdx * SignalLength + pointsIdx];
        coord_t currentIndex = d_multiDirVecChanSignalIndex[dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + pointsIdx];
        coord_t currentFlag = d_sparseMaxFlag[dirVecIdx * SignalLength + pointsIdx];
        coord_t storeLocation = d_MaxScanResult[dirVecIdx * SignalLength + pointsIdx];

        if (currentFlag != 0)
        {
            d_compactMaxValue[dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation] = currentValue;
            d_compactMaxIndex[dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation] = currentIndex;
        }

        if (pointsIdx == SignalLength - 1)
        {
            if (signalDimIdx == 0)
            {
                d_num_extrema_max[dirVecIdx] = storeLocation + 1;
            }
        }
    }
}

template <typename coord_t, typename real_t>
__global__ void select_extrema_min(coord_t* d_sparseMinFlag, real_t* d_current, coord_t* d_multiDirVecChanSignalIndex,
    coord_t* d_MinScanResult, real_t* d_compactMinValue, coord_t* d_compactMinIndex, size_t SignalLength,
    size_t SignalDim, size_t NumDirVector, coord_t* d_num_extrema_min)
{
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pointsIdx < SignalLength)
    {
        real_t currentValue = d_current[signalDimIdx * SignalLength + pointsIdx];
        coord_t currentIndex = d_multiDirVecChanSignalIndex[dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + pointsIdx];
        coord_t currentFlag = d_sparseMinFlag[dirVecIdx * SignalLength + pointsIdx];
        coord_t storeLocation = d_MinScanResult[dirVecIdx * SignalLength + pointsIdx];

        if (currentFlag != 0)
        {
            d_compactMinValue[dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation] = currentValue;
            d_compactMinIndex[dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation] = currentIndex;
        }

        if (pointsIdx == SignalLength - 1)
        {
            if (signalDimIdx == 0)
            {
                d_num_extrema_min[dirVecIdx] = storeLocation + 1;
            }
        }
    }
}

template <typename coord_t, typename real_t>
__global__ void setBoundaryMax(real_t* d_compactMaxValue, coord_t* d_compactMaxIndex, coord_t* d_MaxScanResult, size_t SignalLength, size_t SignalDim, size_t NumDirVector)
{
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = threadIdx.x;

    if ((pointsIdx == 0) && (d_MaxScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4))
    {
        real_t slope_max, t_max;

        coord_t storeLocation_max = d_MaxScanResult[dirVecIdx * SignalLength + pointsIdx];
        coord_t loc_max = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation_max;

        slope_max = (d_compactMaxValue[loc_max + 2] - d_compactMaxValue[loc_max + 1]) / (d_compactMaxIndex[loc_max + 2] - d_compactMaxIndex[loc_max + 1]);
        t_max = d_compactMaxValue[loc_max + 1] - slope_max * (d_compactMaxIndex[loc_max + 1] - d_compactMaxIndex[loc_max]);

        if (t_max > d_compactMaxValue[loc_max])
        {
            d_compactMaxValue[loc_max] = t_max;
        }
    }

    if ((pointsIdx == 1) && (d_MaxScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4))
    {

        real_t slope_max, t_max;

        coord_t storeLocation_max = d_MaxScanResult[dirVecIdx * SignalLength + SignalLength - 1];
        coord_t loc_max = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation_max;

        slope_max = (d_compactMaxValue[loc_max - 1] - d_compactMaxValue[loc_max - 2]) / (d_compactMaxIndex[loc_max - 1] - d_compactMaxIndex[loc_max - 2]);
        t_max = d_compactMaxValue[loc_max - 1] + slope_max * (d_compactMaxIndex[loc_max] - d_compactMaxIndex[loc_max - 1]);

        if (t_max > d_compactMaxValue[loc_max])
        {
            d_compactMaxValue[loc_max] = t_max;
        }
    }
}

template <typename coord_t, typename real_t>
__global__ void setBoundary(real_t* d_compactMaxValue, real_t* d_compactMinValue, coord_t* d_compactMaxIndex, coord_t* d_compactMinIndex, coord_t* d_MaxScanResult, coord_t* d_MinScanResult, size_t SignalLength, size_t SignalDim, size_t NumDirVector)
{
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = threadIdx.x;

    if ((pointsIdx == 0) && (d_MaxScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4) && (d_MinScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4))
    {
        real_t slope_max, t_max, slope_min, t_min;

        coord_t storeLocation_max = d_MaxScanResult[dirVecIdx * SignalLength + pointsIdx];
        coord_t loc_max = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation_max;
        coord_t storeLocation_min = d_MinScanResult[dirVecIdx * SignalLength + pointsIdx];
        coord_t loc_min = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation_min;

        slope_max = (d_compactMaxValue[loc_max + 2] - d_compactMaxValue[loc_max + 1]) / (d_compactMaxIndex[loc_max + 2] - d_compactMaxIndex[loc_max + 1]);
        t_max = d_compactMaxValue[loc_max + 1] - slope_max * (d_compactMaxIndex[loc_max + 1] - d_compactMaxIndex[loc_max]);
        slope_min = (d_compactMinValue[loc_min + 2] - d_compactMinValue[loc_min + 1]) / (d_compactMinIndex[loc_min + 2] - d_compactMinIndex[loc_min + 1]);
        t_min = d_compactMinValue[loc_min + 1] - slope_min * (d_compactMinIndex[loc_min + 1] - d_compactMinIndex[loc_min]);

        if (t_max > d_compactMaxValue[loc_max])
        {
            d_compactMaxValue[loc_max] = t_max;
        }
        if (t_min < d_compactMinValue[loc_min])
        {
            d_compactMinValue[loc_min] = t_min;
        }
    }

    if ((pointsIdx == 1) && (d_MaxScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4) && (d_MinScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4))
    {

        real_t slope_max, t_max, slope_min, t_min;

        coord_t storeLocation_max = d_MaxScanResult[dirVecIdx * SignalLength + SignalLength - 1];
        coord_t loc_max = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation_max;
        coord_t storeLocation_min = d_MinScanResult[dirVecIdx * SignalLength + SignalLength - 1];
        coord_t loc_min = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation_min;

        slope_max = (d_compactMaxValue[loc_max - 1] - d_compactMaxValue[loc_max - 2]) / (d_compactMaxIndex[loc_max - 1] - d_compactMaxIndex[loc_max - 2]);
        t_max = d_compactMaxValue[loc_max - 1] + slope_max * (d_compactMaxIndex[loc_max] - d_compactMaxIndex[loc_max - 1]);
        slope_min = (d_compactMinValue[loc_min - 1] - d_compactMinValue[loc_min - 2]) / (d_compactMinIndex[loc_min - 1] - d_compactMinIndex[loc_min - 2]);
        t_min = d_compactMinValue[loc_min - 1] + slope_min * (d_compactMinIndex[loc_min] - d_compactMinIndex[loc_min - 1]);

        if (t_max > d_compactMaxValue[loc_max])
        {
            d_compactMaxValue[loc_max] = t_max;
        }
        if (t_min < d_compactMinValue[loc_min])
        {
            d_compactMinValue[loc_min] = t_min;
        }
    }
}

template <typename coord_t, typename real_t>
__global__ void setBoundaryMin(real_t* d_compactMinValue, coord_t* d_compactMinIndex, coord_t* d_MinScanResult, size_t SignalLength, size_t SignalDim, size_t NumDirVector)
{
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = threadIdx.x;

    if ((pointsIdx == 0) && (d_MinScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4))
    {
        real_t slope_min, t_min;

        coord_t storeLocation_min = d_MinScanResult[dirVecIdx * SignalLength + pointsIdx];
        coord_t loc_min = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation_min;

        slope_min = (d_compactMinValue[loc_min + 2] - d_compactMinValue[loc_min + 1]) / (d_compactMinIndex[loc_min + 2] - d_compactMinIndex[loc_min + 1]);
        t_min = d_compactMinValue[loc_min + 1] - slope_min * (d_compactMinIndex[loc_min + 1] - d_compactMinIndex[loc_min]);

        if (t_min < d_compactMinValue[loc_min])
        {
            d_compactMinValue[loc_min] = t_min;
        }
    }

    if ((pointsIdx == 1) && (d_MinScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4))
    {

        real_t slope_min, t_min;

        coord_t storeLocation_min = d_MinScanResult[dirVecIdx * SignalLength + SignalLength - 1];
        coord_t loc_min = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation_min;

        slope_min = (d_compactMinValue[loc_min - 1] - d_compactMinValue[loc_min - 2]) / (d_compactMinIndex[loc_min - 1] - d_compactMinIndex[loc_min - 2]);
        t_min = d_compactMinValue[loc_min - 1] + slope_min * (d_compactMinIndex[loc_min] - d_compactMinIndex[loc_min - 1]);

        if (t_min < d_compactMinValue[loc_min])
        {
            d_compactMinValue[loc_min] = t_min;
        }
    }
}

//set boundary (deprecated)
template <typename coord_t, typename real_t>
__global__ void select_extrema_max_boundary(coord_t* d_sparseMaxFlag, real_t* d_current, coord_t* d_multiDirVecChanSignalIndex, coord_t* d_MaxScanResult,
    real_t* d_compactMaxValue, coord_t* d_compactMaxIndex, size_t SignalLength, size_t SignalDim, size_t NumDirVector, coord_t* d_num_extrema_max)
{
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if ((pointsIdx == 0) && (d_MaxScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4))
    {
        real_t slope_max, t_max;
        coord_t storeLocation = d_MaxScanResult[dirVecIdx * SignalLength + pointsIdx];
        coord_t loc = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation;

        real_t num = (d_compactMaxValue[loc + 2] - d_compactMaxValue[loc + 1]);
        coord_t denom = (d_compactMaxIndex[loc + 2] - d_compactMaxIndex[loc + 1]);
        slope_max = num / denom;
        t_max = d_compactMaxValue[loc + 1] - slope_max * (d_compactMaxIndex[loc + 1] - d_compactMaxIndex[loc]);
        if (isinf(t_max))
            printf("inf error-3 in select extrema max \n");

        if (t_max > d_compactMaxValue[loc])
        {
            //if (isinf(t_max))
            //	printf("inf error-2 in select extrema\n");
            d_compactMaxValue[loc] = t_max;
        }
    }

    __threadfence();
    __syncthreads();

    if ((pointsIdx == SignalLength - 1) && (d_MaxScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4))
    {

        real_t slope_max, t_max;
        coord_t storeLocation = d_MaxScanResult[dirVecIdx * SignalLength + pointsIdx];
        coord_t loc = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation;

        //real_t up = d_compactMaxValue[loc - 1] - d_compactMaxValue[loc - 2];
        //real_t down = (real_t)(d_compactMaxIndex[loc - 1] - d_compactMaxIndex[loc - 2]);
        //slope_max = up / (real_t)down;
        slope_max = (d_compactMaxValue[loc - 1] - d_compactMaxValue[loc - 2]) / (d_compactMaxIndex[loc - 1] - d_compactMaxIndex[loc - 2]);
        t_max = d_compactMaxValue[loc - 1] + slope_max * (d_compactMaxIndex[loc] - d_compactMaxIndex[loc - 1]);
        if (isinf(t_max))
            printf("inf error-4 in select extrema max \n");

        if (t_max > d_compactMaxValue[loc])
        {
            d_compactMaxValue[loc] = t_max;
        }
        //if (isinf(d_compactMaxValue[loc]))
        //	printf("inf error-4 in select extrema\n");

    }

}

template <typename coord_t, typename real_t>
__global__ void select_extrema_min_boundary(coord_t* d_sparseMinFlag, real_t* d_current, coord_t* d_multiDirVecChanSignalIndex,
    coord_t* d_MinScanResult, real_t* d_compactMinValue, coord_t* d_compactMinIndex, size_t SignalLength,
    size_t SignalDim, size_t NumDirVector, coord_t* d_num_extrema_min)
{
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if ((pointsIdx == 0) && (d_MinScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4))
    {
        real_t slope_min, t_min;
        coord_t storeLocation = d_MinScanResult[dirVecIdx * SignalLength + pointsIdx];
        coord_t loc = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation;

        slope_min = (d_compactMinValue[loc + 2] - d_compactMinValue[loc + 1]) / (d_compactMinIndex[loc + 2] - d_compactMinIndex[loc + 1]);
        t_min = d_compactMinValue[loc + 1] - slope_min * (d_compactMinIndex[loc + 1] - d_compactMinIndex[loc]);
        if (isinf(t_min))
            printf("inf error-3 in select extrema min \n");

        if (t_min < d_compactMinValue[loc])
        {
            //if (isinf(t_min))
            //	printf("inf error-2 in select extrema \n");
            d_compactMinValue[loc] = t_min;
        }
    }

    __threadfence();
    __syncthreads();

    if ((pointsIdx == SignalLength - 1) && (d_MinScanResult[dirVecIdx * SignalLength + SignalLength - 1] > 4))
    {
        real_t slope_min, t_min;
        coord_t storeLocation = d_MinScanResult[dirVecIdx * SignalLength + pointsIdx];
        coord_t loc = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + storeLocation;

        //real_t up = d_compactMinValue[loc - 1] - d_compactMinValue[loc - 2];
        //real_t down = (real_t)(d_compactMinIndex[loc - 1] - d_compactMinIndex[loc - 2]);
        //slope_min = up / (real_t)down;
        slope_min = (d_compactMinValue[loc - 1] - d_compactMinValue[loc - 2]) / (d_compactMinIndex[loc - 1] - d_compactMinIndex[loc - 2]);
        t_min = d_compactMinValue[loc - 1] + slope_min * (d_compactMinIndex[loc] - d_compactMinIndex[loc - 1]);
        if (isinf(t_min))
            printf("inf error-4 in select extrema min \n");

        if (t_min < d_compactMinValue[loc])
        {
            //if (isinf(t_min))
            //{
            //	printf("inf error-3 in select extrema\n");
            //	printf("d_compactMinIndex[loc - 1]: %d \nd_compactMinIndex[loc - 2]: %d \n", d_compactMinIndex[loc - 1], d_compactMinIndex[loc - 2]);
            //	printf("down: %d\n", down);
            //	printf("slope_min: %f \n", slope_min);
            //}
            d_compactMinValue[loc] = t_min;
        }
    }
}

// for natural boundary conditions
template <typename coord_t, typename real_t>
__global__ void tridiagonal_setup(coord_t* d_num_extrema, coord_t* d_extrema_x, real_t* d_extrema_y, real_t* d_upper_dia, real_t* d_middle_dia, real_t* d_lower_dia, real_t* d_right_dia, size_t SignalLength, size_t SignalDim, size_t NumDirVector) {
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    //int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + pointsIdx;
    const int num_equation = d_num_extrema[dirVecIdx];
    int idxForRight = dirVecIdx * SignalDim * SignalLength + signalDimIdx * num_equation + pointsIdx; // to compact the y value within one direction vector
    if (pointsIdx == 0)
    {
        d_middle_dia[idx] = 1;
        d_upper_dia[idx] = 0;
        d_lower_dia[idx] = 0;
        d_right_dia[idxForRight] = 0; // it used to be idx
    }
    if (pointsIdx == num_equation - 1)
    {
        d_middle_dia[idx] = 1;
        d_lower_dia[idx] = 0;
        d_upper_dia[idx] = 0;
        d_right_dia[idxForRight] = 0;
    }
    if (pointsIdx != 0 && pointsIdx < num_equation - 1)
    {
        d_middle_dia[idx] = 2 * (((d_extrema_x[idx] - d_extrema_x[idx - 1]) + (d_extrema_x[idx + 1] - d_extrema_x[idx])));
        d_upper_dia[idx] = d_extrema_x[idx + 1] - d_extrema_x[idx];
        d_lower_dia[idx] = d_extrema_x[idx] - d_extrema_x[idx - 1];
        d_right_dia[idxForRight] = 3 * ((d_extrema_y[idx + 1] - d_extrema_y[idx]) / (d_extrema_x[idx + 1] - d_extrema_x[idx]) -
            (d_extrema_y[idx] - d_extrema_y[idx - 1]) / (d_extrema_x[idx] - d_extrema_x[idx - 1]));
    }
}

// for not-a-knot boundary conditions
template <typename coord_t, typename real_t>
__global__ void tridiagonal_setup_nak(coord_t* d_num_extrema, coord_t* d_extrema_x, real_t* d_extrema_y, real_t* d_upper_dia, real_t* d_middle_dia, real_t* d_lower_dia, real_t* d_right_dia, size_t SignalLength, size_t SignalDim, size_t NumDirVector) {
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    //int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + pointsIdx;
    const int num_equation = d_num_extrema[dirVecIdx] - 1;
    if (pointsIdx == 0)
    {
        real_t h0 = d_extrema_x[idx + 1] - d_extrema_x[idx];
        real_t h1 = d_extrema_x[idx + 2] - d_extrema_x[idx + 1];
        d_middle_dia[idx] = -h1 * h1 + h0 * h0;
        d_upper_dia[idx] = (h0 + h1) * h1 + 2 * h0 * (h0 + h1);
        d_lower_dia[idx] = 0; // fixed
        d_right_dia[idx] = 3 * (h0 / h1 * (d_extrema_y[idx + 2] - d_extrema_y[idx + 1]) - d_extrema_y[idx + 1] + d_extrema_y[idx]);
    }

    if (pointsIdx == num_equation)
    {
        real_t hn_2 = d_extrema_x[idx - 1] - d_extrema_x[idx - 2];
        real_t hn_1 = d_extrema_x[idx] - d_extrema_x[idx - 1];
        d_middle_dia[idx] = -hn_2 * hn_2 + hn_1 * hn_1;
        d_lower_dia[idx] = (hn_2 + hn_1) * hn_2 + hn_1 * 2 * (hn_2 + hn_1);
        d_upper_dia[idx] = 0; // fixed
        d_right_dia[idx] = 3 * ((d_extrema_y[idx] - d_extrema_y[idx - 1]) - hn_1 / hn_2 * (d_extrema_y[idx - 1] - d_extrema_y[idx - 2]));
    }
    if (pointsIdx != 0 && pointsIdx < num_equation)
    {
        d_middle_dia[idx] = 2 * (((d_extrema_x[idx] - d_extrema_x[idx - 1]) + (d_extrema_x[idx + 1] - d_extrema_x[idx])));
        d_upper_dia[idx] = d_extrema_x[idx + 1] - d_extrema_x[idx];
        d_lower_dia[idx] = d_extrema_x[idx] - d_extrema_x[idx - 1];
        d_right_dia[idx] = 3 * (d_extrema_y[idx + 1] - d_extrema_y[idx]) / (d_extrema_x[idx + 1] - d_extrema_x[idx]) - 3 * (d_extrema_y[idx] - d_extrema_y[idx - 1]) / (d_extrema_x[idx] - d_extrema_x[idx - 1]);
    }
}

template <typename coord_t, typename real_t>
__global__ void spline_coefficients(const real_t* a, real_t* b, real_t* c, real_t* d, coord_t* extrema_points_x, size_t SignalDim, size_t SignalLength, coord_t* d_num_extrema, real_t* solution) {
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + pointsIdx;

    const int num_equation = d_num_extrema[dirVecIdx];
    int idxForSolution = dirVecIdx * SignalDim * SignalLength + signalDimIdx * num_equation + pointsIdx; // to index the compacted sulution matrix, it used to be idx

    if (pointsIdx < num_equation - 1)
    {
        real_t h = extrema_points_x[idx + 1] - extrema_points_x[idx];
        b[idx] = (a[idx + 1] - a[idx]) / h - h * (2 * solution[idxForSolution] + solution[idxForSolution + 1]) / 3;
        d[idx] = (solution[idxForSolution + 1] - solution[idxForSolution]) / (3 * h);
        c[idx] = solution[idxForSolution];
    }

}

template <typename coord_t, typename real_t>
__global__ void interpolate(const real_t* a, real_t* b, real_t* c, real_t* d, coord_t* d_envelopeIndex, real_t* d_envelopeValue, coord_t* d_extremaIndex, size_t SignalLength, coord_t* d_num_extrema, size_t SignalDim, size_t NumDirVector) {
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + pointsIdx;
    int idx2 = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength;

    int num_coefs = d_num_extrema[dirVecIdx];
    size_t num_samples = SignalLength;
    if (pointsIdx < num_samples)// what the hell here? <= or <
    {
        int i = 0;
        int coef_idx = 0;
        int low = 1;
        int high = num_coefs - 1;
        // binary search for coef index
        while (low <= high) {
            int mid = (low + high) / 2;
            if ((pointsIdx > d_extremaIndex[idx2 + mid - 1]) && (pointsIdx <= d_extremaIndex[idx2 + mid])) {
                coef_idx = mid - 1;
                break;
            }
            else if (pointsIdx < d_extremaIndex[idx2 + mid]) {
                high = mid - 1;
            }
            else {
                low = mid + 1;
            }
        }

        coord_t t = d_envelopeIndex[idx] - d_extremaIndex[idx2 + coef_idx];
        d_envelopeValue[idx] = a[idx2 + coef_idx] + (b[idx2 + coef_idx] + (c[idx2 + coef_idx] + d[idx2 + coef_idx] * t) * t) * t;
    }

}

template <typename coord_t, typename real_t>
__global__ void averageUppperLower(real_t* d_meanEnvelope, real_t* d_upperEnvelope, real_t* d_lowerEnvelope, size_t SignalLength, size_t SignalDim, size_t NumDirVector, coord_t* d_num_extrema_max, coord_t* d_num_extrema_min)
{
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + pointsIdx;

    if ((d_num_extrema_max[dirVecIdx] > 3) && (d_num_extrema_min[dirVecIdx] > 3) && (pointsIdx < SignalLength))
    {
        d_meanEnvelope[idx] = (d_upperEnvelope[idx] + d_lowerEnvelope[idx]) / 2.0;
    }
}

template <typename coord_t, typename real_t>
__global__ void averageDirection(real_t* d_current, real_t* d_meanEnvelope, coord_t* d_num_extrema_max, coord_t* d_num_extrema_min, size_t NumDirVector, size_t SignalDim, size_t SignalLength)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < SignalDim * SignalLength)
    {
        real_t value = 0;
        real_t numCompDir = 0;
        for (int i = 0; i < NumDirVector; i++)
        {
            if ((d_num_extrema_max[i] > 3) && (d_num_extrema_min[i] > 3))
            {
                value = value + d_meanEnvelope[i * SignalDim * SignalLength + idx];
                numCompDir = numCompDir + 1;
            }
        }
        d_current[idx] = d_current[idx] - value / numCompDir;
    }
}

template <typename real_t>
__global__ void updateSignal(real_t* d_current, real_t* d_running, size_t SignalDim, size_t SignalLength)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (SignalDim * SignalLength))
    {
        d_running[idx] = d_running[idx] - d_current[idx];
    }
}

__global__ void MatrixMulCUDA(float* C, float* A, float* B, int hA, int wA, int hB, int wB) {
    int colIdx = blockIdx.x * blockDim.x + threadIdx.x; //< 30504
    int rowIdx = blockIdx.y;
    int idx = rowIdx * hA + colIdx;

    if (colIdx < hA)
    {
        double result = 0;
        double tem = 0;

        for (int i = 0; i < wA; i++)
        {
            tem = A[colIdx + i * hA] * B[rowIdx * hB + i];
            result = result + tem;
        }
        C[idx] = (float)result;
    }

}

int getBinSize(char* path)
{
    int  size = 0;
    FILE* fp = fopen(path, "rb");
    if (fp)
    {
        fseek(fp, 0, SEEK_END);
        size = ftell(fp);
        fclose(fp);
    }
    //printf("\npath=%s,size=%d \n", path, size);
    return size;
}

void readBin(char* path, char* buf, int size)
{
    FILE* infile;
    if ((infile = fopen(path, "rb")) == NULL)
    {
        printf("\nCan not open the path: %s \n", path);
        exit(-1);
    }
    fread(buf, sizeof(char), size, infile);
    fclose(infile);
}

void writeBin(char* path, char* buf, int size)
{
    FILE* outfile;
    if ((outfile = fopen(path, "wb")) == NULL)
    {
        printf("\nCan not open the path: %s \n", path);
        exit(-1);
    }
    fwrite(buf, sizeof(char), size, outfile);
    fclose(outfile);
}

__device__ float RadicalInverse(int Base, unsigned i)
{
    float Digit, Radical, Inverse;
    Digit = Radical = 1.0 / Base;
    Inverse = 0.0;
    while (i)
    {
        Inverse += Digit * (i % Base);
        Digit *= Radical;
        i /= Base;
    }
    return Inverse;
}

template <typename T>
__device__ float Sequence2Vector(T* sequence, T* vector, size_t dim, T* tht)
{
    // threads should load data from shared memory, instead of global memory
    int i = 0, j = 0;
    for (i = 0; i < dim; i++)
    {
        sequence[i] = 2 * sequence[i] - 1;
    }
    //T* tht = (T*)malloc((dim - 1) * sizeof(T)); // inefficient here, to be improved
    //T tht[dim - 1];
    for (i = 0; i < dim - 1; i++)
    {
        T tem = 0.0;
        for (j = dim - 1; j > i; j--)
        {
            tem = tem + pow(sequence[j], 2);
        }

        tht[i] = atan2(sqrt(tem), sequence[i]);
    }
    for (i = 0; i < dim; i++)
    {
        if (i == 0)
        {
            vector[i] = 1;
        }
        else
        {
            T tem = 1.0;
            for (j = 0; j < i; j++)
            {
                tem = tem * sin(tht[j]);
            }
            vector[i] = tem;
        }

    }
    for (i = 0; i < dim - 1; i++)
    {
        vector[i] = vector[i] * cos(tht[i]);
    }
}

template <typename T>
__global__ void GenerateHammSeq(int* d_PrimeDataSet, T* d_HammSeq)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (blockIdx.x == 0)
    {
        d_HammSeq[gridDim.x * threadIdx.x + blockIdx.x] = (threadIdx.x + 1.5) / (-d_PrimeDataSet[blockIdx.x]);
    }
    else
    {
        d_HammSeq[gridDim.x * threadIdx.x + blockIdx.x] = RadicalInverse(d_PrimeDataSet[blockIdx.x], threadIdx.x + 1);
    }

}

template <typename T>
__global__ void GenerateDirVec(T* d_HammSeq, T* d_DirectionVector, size_t* d_SignalDim, T* d_tht)
{
    size_t dim = *d_SignalDim;
    Sequence2Vector(&d_HammSeq[dim * threadIdx.x], &d_DirectionVector[dim * threadIdx.x], dim, &d_tht[(dim - 1) * threadIdx.x]);
}

int isPrimes(int n)
{
    int i;
    int flag = 0;
    for (i = 2; i <= n / 2; ++i)
    {
        if (n % i == 0)
        {
            flag = 1;
            break;
        }
    }
    return flag;
}

template <typename T>
void GenerateDirectionVector(size_t SignalDim, size_t NumDirVector, T* d_DirectionVector)
{
    // generate primes dataset on cpu
    size_t NumPrimes = SignalDim;
    size_t PrimesCounter = 1;
    int PotentialPrimes = 2;
    int* h_PrimeDataSet = (int*)malloc(NumPrimes * sizeof(int));
    while (PrimesCounter < NumPrimes)
    {
        if (!isPrimes(PotentialPrimes))
        {
            h_PrimeDataSet[PrimesCounter] = PotentialPrimes;
            PrimesCounter++;
        }
        PotentialPrimes++;
    }
    h_PrimeDataSet[0] = -(int)NumDirVector;

    // generate hammersley sequence using primes dataset
    int* d_PrimeDataSet = NULL;
    T* d_HammSeq = NULL;
    hipMalloc((void**)&d_HammSeq, SignalDim * NumDirVector * sizeof(T));
    hipMalloc((void**)&d_PrimeDataSet, NumPrimes * sizeof(int));
    hipMemcpy(d_PrimeDataSet, h_PrimeDataSet, NumPrimes * sizeof(int), hipMemcpyHostToDevice);

    dim3 block_dim = NumDirVector;
    dim3 grid_dim = SignalDim;
    GenerateHammSeq << <grid_dim, block_dim >> > (d_PrimeDataSet, d_HammSeq);

    // convert hammersley sequence to direction vector
    size_t* d_SignalDim = NULL;
    hipMalloc((void**)&d_SignalDim, sizeof(size_t));
    hipMemcpy(d_SignalDim, &SignalDim, sizeof(size_t), hipMemcpyHostToDevice);

    T* d_tht = NULL;
    hipMalloc((void**)&d_tht, (SignalDim - 1) * NumDirVector * sizeof(T));
    hipMemset(d_tht, 0, (SignalDim - 1) * NumDirVector * sizeof(T));

    block_dim = NumDirVector;
    grid_dim = 1;
    GenerateDirVec << <grid_dim, block_dim >> > (d_HammSeq, d_DirectionVector, d_SignalDim, d_tht);

}

template <typename coord_t, typename real_t>
double memd(size_t SignalDim, size_t SignalLength, size_t NumDirVector, size_t num_IMFs, size_t max_iter, coord_t* d_x, real_t* d_y, real_t* d_DirectionVectors, real_t* d_IMFs)
{
    //===============load data===============
    real_t* d_current = NULL;
    hipMalloc((void**)&d_current, SignalDim * SignalLength * sizeof(real_t));
    real_t* d_running = NULL;
    hipMalloc((void**)&d_running, SignalDim * SignalLength * sizeof(real_t));

    //===============preparation for projection===============
    real_t* d_ProjectSignals = NULL;
    hipMalloc((void**)&d_ProjectSignals, NumDirVector * SignalLength * sizeof(real_t));
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    float alpha = 1;
    float beta = 0;

    //===============preparation for extreme points detection===============
    coord_t* d_sparseFlag;
    hipMalloc((void**)&d_sparseFlag, NumDirVector * SignalLength * sizeof(coord_t));

    coord_t* d_multiProjectSignalIndex = NULL;
    hipMalloc((void**)&d_multiProjectSignalIndex, NumDirVector * SignalLength * sizeof(coord_t));
    for (int i = 0; i < NumDirVector; i++)
    {
        int head = i * SignalLength;
        hipMemcpy((d_multiProjectSignalIndex + head), d_x, SignalLength * sizeof(coord_t), hipMemcpyDeviceToDevice);
    }

    //===============preparation for prefix sum===============
    coord_t* d_ScanResult;
    hipMalloc((void**)&d_ScanResult, NumDirVector * SignalLength * sizeof(coord_t));
    int* d_sums, * d_incr;
    hipMalloc((void**)&d_sums, SignalLength * sizeof(int));
    hipMalloc((void**)&d_incr, SignalLength * sizeof(int));

    //===============preparation for extreme points select===============
    real_t* d_compactValue;
    coord_t* d_compactIndex;
    coord_t* d_num_extrema_max, * d_num_extrema_min;

    hipMalloc((void**)&d_compactValue, NumDirVector * SignalDim * SignalLength * sizeof(real_t));
    hipMalloc((void**)&d_compactIndex, NumDirVector * SignalDim * SignalLength * sizeof(coord_t));
    hipMalloc((void**)&d_num_extrema_max, NumDirVector * sizeof(coord_t));
    hipMalloc((void**)&d_num_extrema_min, NumDirVector * sizeof(coord_t));

    //===============preparation for tridiagonal setting===============
    real_t* d_upperDia = NULL, * d_middleDia = NULL, * d_lowerDia = NULL, * d_right = NULL;

    hipMalloc((void**)&d_upperDia, NumDirVector * SignalDim * SignalLength * sizeof(real_t));
    hipMalloc((void**)&d_middleDia, NumDirVector * SignalDim * SignalLength * sizeof(real_t));
    hipMalloc((void**)&d_lowerDia, NumDirVector * SignalDim * SignalLength * sizeof(real_t));
    hipMalloc((void**)&d_right, NumDirVector * SignalDim * SignalLength * sizeof(real_t));

    //===============preparation for tridiagonal solving===============
    real_t* d_solutionGtsv = NULL;
    hipMalloc((void**)&d_solutionGtsv, NumDirVector * SignalDim * SignalLength * sizeof(real_t));

    real_t* currentUpperDia = NULL;
    real_t* currentMiddleDia = NULL;
    real_t* currentLowerDia = NULL;
    real_t* currentRightDia = NULL;
    real_t* currentSolution = NULL;
    coord_t* h_num_extrema = (coord_t*)malloc(NumDirVector * sizeof(coord_t));

    hipsparseHandle_t handle_sparse;
    hipsparseCreate(&handle_sparse);
    size_t* buffer_size = (size_t*)malloc(sizeof(size_t));
    real_t* buffer = NULL;
    hipMalloc(&buffer, 1024 * 1024 * sizeof(real_t));

    //===============preparation for spline coefficients calculation===============
    real_t* d_b = NULL, * d_d = NULL, * d_c = NULL;

    //===============preparation for interpolate values===============
    coord_t* d_multiDirVecChanSignalIndex = NULL;
    hipMalloc((void**)&d_multiDirVecChanSignalIndex, NumDirVector * SignalDim * SignalLength * sizeof(coord_t));

    for (int i = 0; i < NumDirVector * SignalDim; i++)
    {
        int head = i * SignalLength;
        hipMemcpy((d_multiDirVecChanSignalIndex + head), d_x, SignalLength * sizeof(coord_t), hipMemcpyDeviceToDevice);
    }

    real_t* d_envelopeVauleMax = NULL, * d_envelopeVauleMin = NULL;
    hipMalloc((void**)&d_envelopeVauleMax, NumDirVector * SignalDim * SignalLength * sizeof(real_t));
    hipMalloc((void**)&d_envelopeVauleMin, NumDirVector * SignalDim * SignalLength * sizeof(real_t));

    //===============preparation for averaging upper and lower===============
    real_t* d_meanEnvelope = NULL;
    hipMalloc((void**)&d_meanEnvelope, NumDirVector * SignalDim * SignalLength * sizeof(real_t));

    //===============preparation for MULTI-STREAM===============
    //const int num_streams = 2;
    //hipStream_t streams[num_streams];
    //hipStreamCreate(&streams[1]);
    //hipStreamCreate(&streams[2]);

    //===============start looping===============
    hipMemcpy(d_running, d_y, SignalDim * SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);
    double start_time = omp_get_wtime();
    for (size_t i = 0; i < num_IMFs - 1; ++i)
    {
        hipMemcpy(d_current, d_running, SignalDim * SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);
        for (size_t j = 0; j < max_iter; ++j)
        {
            //==================multivariate signal projection============
            hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, SignalLength, NumDirVector, SignalDim,
                &alpha, d_current, SignalLength, d_DirectionVectors, SignalDim,
                &beta, d_ProjectSignals, SignalLength);

            //dim3 threads(1024);
            //dim3 grid(SignalLength / threads.x + 1, NumDirVector);
            //MatrixMulCUDA <<<grid, threads >>> (d_ProjectSignals, d_current, d_DirectionVectors, SignalLength, SignalDim, SignalDim, NumDirVector);

            //==================extreme points detection max============
            dim3 blockDimShfl(256);
            size_t offset = (blockDimShfl.x / 32 - 1) * 2;
            dim3 gridDimShfl(SignalLength / (blockDimShfl.x - offset) + (SignalLength % (blockDimShfl.x - offset) == 0 ? 0 : 1), NumDirVector);
            hipMemset(d_sparseFlag, 0, NumDirVector * SignalLength * sizeof(coord_t));
            find_extrema_shfl_max << <gridDimShfl, blockDimShfl >> > (d_multiProjectSignalIndex, d_ProjectSignals, d_sparseFlag, SignalLength);

            //==================prefix scan max============
            for (size_t k = 0; k < NumDirVector; k++)
            {
                int offset = k * SignalLength;
                if (SignalLength > ELEMENTS_PER_BLOCK) {
                    scanLargeDeviceArray(d_ScanResult + offset, d_sparseFlag + offset, SignalLength, 1, d_sums, d_incr);
                }
                else {
                    scanSmallDeviceArray(d_ScanResult + offset, d_sparseFlag + offset, SignalLength, 1);
                }
            }

            //==================extreme points select max============
            dim3 blockDimSelectExtrema(256);
            dim3 gridDimSelectExtrema(SignalLength / blockDimSelectExtrema.x + (SignalLength % blockDimSelectExtrema.x == 0 ? 0 : 1), SignalDim, NumDirVector);
            select_extrema_max << <gridDimSelectExtrema, blockDimSelectExtrema >> > (d_sparseFlag, d_current, d_multiDirVecChanSignalIndex,
                d_ScanResult, d_compactValue, d_compactIndex, SignalLength, SignalDim, NumDirVector, d_num_extrema_max);

            dim3 blockDimSetBoundary(2);
            dim3 gridDimSetBoundary(1, SignalDim, NumDirVector);
            setBoundaryMax << <gridDimSetBoundary, blockDimSetBoundary >> > (d_compactValue, d_compactIndex,
                d_ScanResult, SignalLength, SignalDim, NumDirVector);

            //==================set up tridiagonal matrix max============
            dim3 blockDimTriSet(256);
            dim3 gridDimTriSet(SignalLength / blockDimTriSet.x + (SignalLength % blockDimTriSet.x == 0 ? 0 : 1), SignalDim, NumDirVector); // too much idle threads
            tridiagonal_setup << <gridDimTriSet, blockDimTriSet >> > (d_num_extrema_max, d_compactIndex, d_compactValue,
                d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength, SignalDim, NumDirVector);

            //==================solve tridiagonal matrix max============
            hipMemcpy(d_solutionGtsv, d_right, NumDirVector * SignalDim * SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);
            hipMemcpy(h_num_extrema, d_num_extrema_max, NumDirVector * sizeof(coord_t), hipMemcpyDeviceToHost);
            for (size_t k = 0; k < NumDirVector; k++)
            {
                currentUpperDia = d_upperDia + k * SignalLength * SignalDim;
                currentMiddleDia = d_middleDia + k * SignalLength * SignalDim;
                currentLowerDia = d_lowerDia + k * SignalLength * SignalDim;
                currentRightDia = d_right + k * SignalLength * SignalDim;
                currentSolution = d_solutionGtsv + k * SignalLength * SignalDim;

                hipsparseSgtsv2_nopivot_bufferSizeExt(handle_sparse, h_num_extrema[k], SignalDim, currentLowerDia, currentMiddleDia, currentUpperDia, currentRightDia, h_num_extrema[k], buffer_size);
                //hipMalloc(&buffer, *buffer_size); //to be optimized
                hipsparseSgtsv2_nopivot(handle_sparse, h_num_extrema[k], SignalDim, currentLowerDia, currentMiddleDia, currentUpperDia, currentSolution, h_num_extrema[k], buffer);
                //hipFree(buffer);
            }

            //==================compute spline coefficients max============
            dim3 blockDimSplineCoe(256);
            dim3 gridDimSplineCoe(SignalLength / blockDimSplineCoe.x + (SignalLength % blockDimSplineCoe.x == 0 ? 0 : 1), SignalDim, NumDirVector); // too much idle threads
            //hipMemset(d_upperDia, 0, NumDirVector * SignalDim * SignalLength * sizeof(real_t)); // to replace d_b
            //hipMemset(d_middleDia, 0, NumDirVector * SignalDim * SignalLength * sizeof(real_t)); // to replace d_c
            //hipMemset(d_lowerDia, 0, NumDirVector * SignalDim * SignalLength * sizeof(real_t)); // to replace d_d
            spline_coefficients << <gridDimSplineCoe, blockDimSplineCoe >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_compactIndex, SignalDim, SignalLength, d_num_extrema_max, d_solutionGtsv);

            //==================interpolate values max============
            dim3 blockDimInterpolate(256);
            dim3 gridDimInterpolate(SignalLength / blockDimInterpolate.x + (SignalLength % blockDimInterpolate.x == 0 ? 0 : 1), SignalDim, NumDirVector);
            interpolate << <gridDimInterpolate, blockDimInterpolate >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_multiDirVecChanSignalIndex, d_envelopeVauleMax, d_compactIndex, SignalLength, d_num_extrema_max, SignalDim, NumDirVector);

            //==================extreme points detection min============
            hipMemset(d_sparseFlag, 0, NumDirVector * SignalLength * sizeof(coord_t));
            find_extrema_shfl_min << <gridDimShfl, blockDimShfl >> > (d_multiProjectSignalIndex, d_ProjectSignals, d_sparseFlag, SignalLength);

            //==================prefix scan min============
            for (size_t k = 0; k < NumDirVector; k++)
            {
                int offset = k * SignalLength;
                if (SignalLength > ELEMENTS_PER_BLOCK) {
                    scanLargeDeviceArray(d_ScanResult + offset, d_sparseFlag + offset, SignalLength, 1, d_sums, d_incr);
                }
                else {
                    scanSmallDeviceArray(d_ScanResult + offset, d_sparseFlag + offset, SignalLength, 1);
                }
            }

            //==================extreme points select min============
            select_extrema_min << <gridDimSelectExtrema, blockDimSelectExtrema >> > (d_sparseFlag, d_current, d_multiDirVecChanSignalIndex,
                d_ScanResult, d_compactValue, d_compactIndex, SignalLength, SignalDim, NumDirVector, d_num_extrema_min);

            setBoundaryMin << <gridDimSetBoundary, blockDimSetBoundary >> > (d_compactValue, d_compactIndex,
                d_ScanResult, SignalLength, SignalDim, NumDirVector);

            //==================set up tridiagonal matrix min============
            tridiagonal_setup << <gridDimTriSet, blockDimTriSet >> > (d_num_extrema_min, d_compactIndex, d_compactValue,
                d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength, SignalDim, NumDirVector);

            //==================solve tridiagonal matrix min============
            hipMemcpy(d_solutionGtsv, d_right, NumDirVector * SignalDim * SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);
            hipMemcpy(h_num_extrema, d_num_extrema_min, NumDirVector * sizeof(coord_t), hipMemcpyDeviceToHost);

            for (size_t k = 0; k < NumDirVector; k++)
            {
                currentUpperDia = d_upperDia + k * SignalLength * SignalDim;
                currentMiddleDia = d_middleDia + k * SignalLength * SignalDim;
                currentLowerDia = d_lowerDia + k * SignalLength * SignalDim;
                currentRightDia = d_right + k * SignalLength * SignalDim;
                currentSolution = d_solutionGtsv + k * SignalLength * SignalDim;

                hipsparseSgtsv2_nopivot_bufferSizeExt(handle_sparse, h_num_extrema[k], SignalDim, currentLowerDia, currentMiddleDia, currentUpperDia, currentRightDia, h_num_extrema[k], buffer_size);
                //hipMalloc(&buffer, *buffer_size); //to be optimized
                hipsparseSgtsv2_nopivot(handle_sparse, h_num_extrema[k], SignalDim, currentLowerDia, currentMiddleDia, currentUpperDia, currentSolution, h_num_extrema[k], buffer);
                //hipFree(buffer);
            }

            //==================compute spline coefficients min============
            //hipMemset(d_upperDia, 0, NumDirVector* SignalDim* SignalLength * sizeof(real_t)); // to replace d_b
            //hipMemset(d_middleDia, 0, NumDirVector* SignalDim* SignalLength * sizeof(real_t)); // to replace d_c
            //hipMemset(d_lowerDia, 0, NumDirVector* SignalDim* SignalLength * sizeof(real_t)); // to replace d_d
            spline_coefficients << <gridDimSplineCoe, blockDimSplineCoe >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_compactIndex, SignalDim, SignalLength, d_num_extrema_min, d_solutionGtsv);

            //==================interpolate values min============
            interpolate << <gridDimInterpolate, blockDimInterpolate >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_multiDirVecChanSignalIndex, d_envelopeVauleMin, d_compactIndex, SignalLength, d_num_extrema_min, SignalDim, NumDirVector);

            //==================average upper and lower============
            dim3 blockDimMeanEnvelope(256);
            dim3 gridDimMeanEnvelope(SignalLength / blockDimInterpolate.x + (SignalLength % blockDimInterpolate.x == 0 ? 0 : 1), SignalDim, NumDirVector);
            averageUppperLower << <gridDimMeanEnvelope, blockDimMeanEnvelope >> > (d_meanEnvelope, d_envelopeVauleMax, d_envelopeVauleMin, SignalLength, SignalDim, NumDirVector, d_num_extrema_max, d_num_extrema_min);

            //==================average each direction and update d_current============
            dim3 blockDimMeanDir(256);
            dim3 gridDimMeanDir(SignalDim * SignalLength / blockDimMeanDir.x + (SignalDim * SignalLength % blockDimMeanDir.x == 0 ? 0 : 1));
            averageDirection << <gridDimMeanDir, blockDimMeanDir >> > (d_current, d_meanEnvelope, d_num_extrema_max, d_num_extrema_min, NumDirVector, SignalDim, SignalLength);

            //=============================TEST============================
            //real_t* h_current_new = (real_t*)malloc(SignalDim * SignalLength * sizeof(real_t));
            //hipMemcpy(h_current_new, d_current, SignalDim * SignalLength * sizeof(real_t), hipMemcpyDeviceToHost);
            //char test_file[] = "C:\\Users\\BRAIN-1\\source\\repos\\EMD project\\MEMD_Algorithm\\MEMD\\test_1.bin";
            //writeBin(test_file, (char*)h_test, SignalDim * SignalLength * sizeof(real_t));
            //printf("IMF: %d, Iter: %d, averageDir %f \n", i, j, h_current_new[0]);
            //=============================TEST============================
        }
        hipMemcpy(&d_IMFs[i * SignalDim * SignalLength], d_current, SignalDim * SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);

        dim3 blockDimUpdateSignal(256);
        dim3 gridDimUpdateSignal(SignalDim * SignalLength / blockDimUpdateSignal.x + (SignalDim * SignalLength % blockDimUpdateSignal.x == 0 ? 0 : 1));
        updateSignal << <gridDimUpdateSignal, blockDimUpdateSignal >> > (d_current, d_running, SignalDim, SignalLength);
        //printf("%s, IMF: %d, updateSignal \n", hipGetErrorString(hipGetLastError()), i);

    }
    double end_time = omp_get_wtime();
    double diff = end_time - start_time;
    hipMemcpy(&d_IMFs[(num_IMFs - 1) * SignalDim * SignalLength], d_running, SignalDim * SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);

    //free all the CPU and GPU memory here
    hipFree(d_current);
    hipFree(d_running);
    hipFree(d_ProjectSignals);
    hipFree(d_sparseFlag);
    hipFree(d_multiProjectSignalIndex);
    hipFree(d_ScanResult);
    hipFree(d_sums);
    hipFree(d_incr);
    hipFree(d_compactValue);
    hipFree(d_compactIndex);
    hipFree(d_num_extrema_max);
    hipFree(d_num_extrema_min);
    hipFree(d_upperDia);
    hipFree(d_middleDia);
    hipFree(d_lowerDia);
    hipFree(d_right);
    hipFree(d_solutionGtsv);
    //hipFree(d_b);
    //hipFree(d_d);
    //hipFree(d_c);
    hipFree(d_multiDirVecChanSignalIndex);
    hipFree(d_envelopeVauleMax);
    hipFree(d_envelopeVauleMin);
    hipFree(d_meanEnvelope);
    free(h_num_extrema);
    hipFree(buffer);
    return diff;
}

void runBenchmark()
{
    size_t num_IMFs = 8;
    size_t max_iter = 10;

    //char filePath1[] = "./S6.bin";
    //char filePath2[] = "./S16.bin";
    char filePath1[] = "./InputData_32CH_244032.bin";
    char filePath2[] = "./InputData_64CH_244032.bin";
    char filePath3[] = "./InputData_128CH_244032.bin";
    char* allFilePath[] = { filePath1, filePath2, filePath3 };
    //char* allFilePath[] = { filePath1, filePath2 };
    int signalDimension[3] = { 32, 64, 128 };
    int directionVectors[3] = { 64, 128, 256 };
    //int signalLength[1] = { 2049 };
    int signalLength[118] = { 2049, 4097, 6145, 8193, 10241, 12289, 14337, 16385, 18433, 20481,
                            22529, 24577, 26625, 28673, 30721, 32769, 34817, 36865, 38913, 40961,
                            43009, 45057, 47105, 49153,	51201, 53249, 55297, 57345,	59393, 61441,
                            63489, 65537, 67585, 69633,	71681, 73729, 75777, 77825,	79873, 81921,
                            83969, 86017, 88065, 90113,	92161, 94209, 96257, 98305,	100353, 102401,
                            104449,	106497,	108545, 110593, 112641, 114689,	116737,	118785,	120833,
                            122881,	124929,	126977,	129025,	131073,	133121,	135169,	137217,	139265,
                            141313,	143361, 145409, 147457, 149505,	151553,	153601,	155649,	157697,
                            159745,	161793,	163841,	165889,	167937,	169985,	172033,	174081, 176129,
                            178177,	180225,	182273,	184321,	186369,	188417,	190465,	192513,	194561,
                            196609,	198657,	200705,	202753,	204801,	206849,	208897,	210945,	212993,
                            215041,	217089,	219137, 221185,	223233,	225281,	227329,	229377,	231425,
                            233473,	235521,	237569,	239617,	241665 };

    for (int fileIdx = 1; fileIdx < 2; fileIdx++) // iterate channel: 32, 64, 128
    {
        char* filePath = allFilePath[fileIdx];
        size_t nbytes = getBinSize(filePath); // in bytes
        const size_t SignalLength = nbytes / sizeof(float) / signalDimension[fileIdx];
        size_t oneChannelNbytes_coord = SignalLength * sizeof(int); // one dimention of the multivariate input signal
        size_t oneChannelNbytes_real = SignalLength * sizeof(float);

        // allocate array on host
        float* h_y;
        int* h_x;
        h_x = (int*)malloc(oneChannelNbytes_coord);
        h_y = (float*)malloc(oneChannelNbytes_real * signalDimension[fileIdx]);
        // load data
        char* buf = (char*)malloc(oneChannelNbytes_real * signalDimension[fileIdx]);
        readBin(filePath, buf, oneChannelNbytes_real * signalDimension[fileIdx]);
        h_y = (float*)buf;
        // generate data index
        for (int i = 0; i < SignalLength; i++) {
            h_x[i] = i;
        }

        for (int dirVecIdx = 1; dirVecIdx < 2; dirVecIdx++) // iterate direction vectors: 64, 128, 256
        {
            for (int sigLenIdx = 2; sigLenIdx < 3; sigLenIdx++) // iterate signal length
            {
                float* d_y;
                int* d_x;
                size_t oneChannelNbytes_coord = signalLength[sigLenIdx] * sizeof(int); // one dimention of the multivariate input signal
                size_t oneChannelNbytes_real = signalLength[sigLenIdx] * sizeof(float);
                // allocate array on device
                hipMalloc((void**)&d_x, oneChannelNbytes_coord);
                hipMalloc((void**)&d_y, oneChannelNbytes_real * signalDimension[fileIdx]);
                // copy data to device
                hipMemcpy(d_x, h_x, oneChannelNbytes_coord, hipMemcpyHostToDevice);
                for (int sigDimIdx = 0; sigDimIdx < signalDimension[fileIdx]; sigDimIdx++)
                {
                    float* current_h_y = h_y + sigDimIdx * SignalLength;
                    float* current_d_y = d_y + sigDimIdx * signalLength[sigLenIdx];
                    hipMemcpy(current_d_y, current_h_y, oneChannelNbytes_real, hipMemcpyHostToDevice);
                }

                // direction vector generation
                float* d_DirectionVectors = NULL;
                hipMalloc(&d_DirectionVectors, signalDimension[fileIdx] * directionVectors[dirVecIdx] * sizeof(float));
                GenerateDirectionVector(signalDimension[fileIdx], directionVectors[dirVecIdx], d_DirectionVectors);

                float* d_IMFs = NULL;
                hipMalloc(&d_IMFs, num_IMFs * signalDimension[fileIdx] * signalLength[sigLenIdx] * sizeof(float));// each IMF is a multivariate IMF
                double time = memd(signalDimension[fileIdx], signalLength[sigLenIdx], directionVectors[dirVecIdx], num_IMFs, max_iter, d_x, d_y, d_DirectionVectors, d_IMFs);
                printf("Channel: %d, Length: %d, DirVec: %d, Time: %f \n", signalDimension[fileIdx], signalLength[sigLenIdx], directionVectors[dirVecIdx], time);

                float* IMFs = (float*)malloc(num_IMFs * signalDimension[fileIdx] * signalLength[sigLenIdx] * sizeof(float));
                hipMemcpy(IMFs, d_IMFs, num_IMFs * signalDimension[fileIdx] * signalLength[sigLenIdx] * sizeof(float), hipMemcpyDeviceToHost);
                char IMFs_file[] = "./mode_matrix.bin";
                writeBin(IMFs_file, (char*)IMFs, num_IMFs * signalDimension[fileIdx] * signalLength[sigLenIdx] * sizeof(float));

                //free all the mem
                hipFree(d_y);
                hipFree(d_x);
                hipFree(d_DirectionVectors);
                hipFree(d_IMFs);
            }
        }
    }
}

int main()
{
    //runBenchmark();

    //=============================Single running=============================

    configuration for the input signal
    size_t SignalDim = 32;
    size_t NumDirVector = 64;
    size_t num_IMFs = 9;
    size_t max_iter = 10;

    float* h_y, * d_y;
    int* h_x, * d_x;

    // get data size
    //char* filePathInput;
    //filePathInput = argv[5];
    char filePathInput[] = "./EEGsample_32CH_30504.bin";
    size_t nbytes = getBinSize(filePathInput); // in bytes
    const size_t SignalLength = nbytes / sizeof(float) / SignalDim;
    size_t oneChannelNbytes_coord = SignalLength * sizeof(int); // one dimention of the multivariate input signal
    size_t oneChannelNbytes_real = SignalLength * sizeof(float);

    // allocate array on host
    h_x = (int*)malloc(oneChannelNbytes_coord);
    h_y = (float*)malloc(oneChannelNbytes_real * SignalDim);

    // allocate array on device
    hipMalloc((void**)&d_x, oneChannelNbytes_coord);
    hipMalloc((void**)&d_y, oneChannelNbytes_real * SignalDim);

    // load data
    char* buf = (char*)malloc(oneChannelNbytes_real * SignalDim);
    readBin(filePathInput, buf, oneChannelNbytes_real * SignalDim);
    h_y = (float*)buf;

    // generate data index
    for (size_t i = 0; i < SignalLength; i++) {
        h_x[i] = (int)i;
    }

    // copy data to device
    hipMemcpy(d_x, h_x, oneChannelNbytes_coord, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, oneChannelNbytes_real * SignalDim, hipMemcpyHostToDevice);

    // direction vector generation
    float* d_DirectionVectors = NULL;
    hipMalloc(&d_DirectionVectors, SignalDim * NumDirVector * sizeof(float));
    GenerateDirectionVector(SignalDim, NumDirVector, d_DirectionVectors);

    //memd processing
    float* d_IMFs = NULL;
    hipMalloc(&d_IMFs, num_IMFs * SignalDim * SignalLength * sizeof(float));// each IMF is a multivariate IMF
    float* IMFs = (float*)malloc(num_IMFs * SignalDim * SignalLength * sizeof(float));

    memd(SignalDim, SignalLength, NumDirVector, num_IMFs, max_iter, d_x, d_y, d_DirectionVectors, d_IMFs);

    hipMemcpy(IMFs, d_IMFs, num_IMFs * SignalDim * SignalLength * sizeof(float), hipMemcpyDeviceToHost);
    //char* IMFs_file = argv[6];
    char IMFs_file[] = "./mode_matrix.bin";
    writeBin(IMFs_file, (char*)IMFs, num_IMFs * SignalDim * SignalLength * sizeof(float));

}